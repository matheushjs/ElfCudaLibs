#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

/*
 * MACROS for building programs BUILD_TEST mode or BUILD_USER mode
 *    TEST - when run, the program will execute the batch of tests we defined
 *    USER - when run, the program will count collisions for a vector of size provided as program argument
 * If nothing is specified, BUILD_USER is assumed.
 */
#ifdef BUILD_USER
	#define BUILD_USER 1
#else
	#define BUILD_USER 0
#endif
#ifdef BUILD_TEST
	#define BUILD_TEST 1
#else
	#define BUILD_TEST 0
#endif


/*
 * MACROS for defining which collision count method to test.
 */
#ifdef SEQUENTIAL_QUADRATIC
	#define METHOD 0
#endif
#ifdef SEQUENTIAL_LINEAR
	#define METHOD 1
#endif
#ifdef NSTEPS_SINGLEROW
	#define METHOD 2
#endif
#ifdef NSTEPS_MULTIROW
	#define METHOD 3
#endif
#ifdef HALFSTEPS_SINGLEROW
	#define METHOD 4
#endif
#ifdef SINGLESTEPS_ALLTHREADS
	#define METHOD 5
#endif
#ifdef SINGLESTEPS_HALFTHREADS
	#define METHOD 6
#endif
#ifndef METHOD // If no method is defined
	#define METHOD 0
#endif

/*
 * Include collision count method as defined by macro METHOD
 */
#if METHOD == 0
	#include "Sequential_Quadratic/test.cuh"
#elif METHOD == 1
	#include "Sequential_Linear/test.cuh"
#elif METHOD == 2
	#include "NSteps_SingleRow/test.cuh"
#elif METHOD == 3
	#include "NSteps_MultiRow/test.cuh"
#elif METHOD == 4
	#include "HalfSteps_SingleRow/test.cuh"
#elif METHOD == 5
	#include "SingleSteps_AllThreads/test.cuh"
#elif METHOD == 6
	#include "SingleSteps_HalfThreads/test.cuh"
#else
	#error "Fix method mate."
#endif

int3 dummy[] = {
		{0, 0, 0}, // 0
		{0, 0, 0}, // 1
		{0, 0, 1}, // 2
		{0, 0, 2}, // 3
		{0, 0, 3}, // 4
		{0, 1, 3}, // 5
		{0, 2, 3}, // 6
		{0, 2, 2}, // 7
		{0, 2, 1}, // 8
		{0, 2, 0}, // 9
		{0, 1, 0}, // 10
		{0, 0, 0}, // 11
		{-1,0, 0}, // 12
		{-2,0, 0}, // 13
		{-2,-1,0}, // 14
		{-1,-1,0}, // 15
		{0 ,-1,0}, // 16
		{0, 0, 0}, // 17
		{0, 0, 0}  // 18
}; // There are 5 {0,0,0}, meaning 4 + 3 + 2 + 1 = 10 collisions

int3 dummy2[] = {
	{0, 0, 0},
	{0, 0, 0},
	{1, 0, 0},
	{1, 0, 0}
}; // 2 collision

int3 *create_vector(int size){
	int i;
	int3 *result = (int3 *) malloc(sizeof(int3) * size);

	for(i = 0; i < size; i++){
		result[i].x = 0;
		result[i].y = 0;
		result[i].z = (i % (size/2)) * 4 - size;

		// Randomize due to caching effects
		// result[i].x = rand()%(2*size) - size;
		// result[i].y = rand()%(2*size) - size;
		// result[i].z = rand()%(2*size) - size;
	}

	printf("Collisions expected: %d\n", size/2);

	return result;
}

// Creates random vector with size/2 collisions
int3 *random_vector(int size){
	int i;
	int3 *result = (int3 *) malloc(sizeof(int3) * size);

	// Generate beads with size/2 collisions
	for(i = 0; i < size; i += 2){
		result[i].x = (i*2) - size; // Guarantees different positions among pairs of beads
		result[i].y = rand()%(2*size) - size;
		result[i].z = rand()%(2*size) - size;

		// Next bead is the same as previous
		result[i+1] = result[i];
	}

	// Randomize bead positions
	for(i = 0; i < size; i++){
		int a = rand()%size;
		int b = rand()%size;

		int3 aux = result[a];
		result[a] = result[b];
		result[b] = aux;
	}

	printf("Collisions expected: %d\n", size/2);

	return result;
}

int3 *sequential_vector(int size){
	int3 *result = (int3 *) malloc(sizeof(int3) * size);

	for(int i = 0; i < size; i++){
		result[i] = (int3) {i, i, i};
	}

	return result;
}

void t1(){
	int dummySize = sizeof(dummy) / sizeof(int3);
	test_count(dummy, dummySize, 1);
}

void t2(int vecSize, int iters){
	// int vecSize = 1000;
	// int iters = 10000;

	int3 *vec = random_vector(vecSize);

	test_count(vec, vecSize, iters);
	free(vec);
}

void t3(){
	int size = 16 * 1024;
	int gold, res;

	// First we create a vector where neighbors have collisions
	int3 *vec = sequential_vector(size);
	for(int i = 0; i < size; i += 2){
		vec[i] = vec[i+1];
	}
	gold = size/2;
	res = test_count(vec, size, 1);
	printf("Expected: %d\n", gold);
	printf("Got:      %d\n", res);
	free(vec);
	printf("Size %d: %s\n\n\n", size, gold == res ? "SUCCESS!" : "FAILURE");

	// Then we create a vector where all elements are colliding
	vec = sequential_vector(size);
	for(int i = 0; i < size; i++){
		vec[i] = vec[0];
	}
	gold = size * (size - 1) / 2;
	res = test_count(vec, size, 1);
	printf("Expected: %d\n", gold);
	printf("Got:      %d\n", res);
	free(vec);
	printf("Size %d: %s\n\n\n", size, gold == res ? "SUCCESS!" : "FAILURE");

	// Finally, no collisions at all
	vec = sequential_vector(size);
	gold = 0;
	res = test_count(vec, size, 1);
	printf("Expected: %d\n", gold);
	printf("Got:      %d\n", res);
	free(vec);
	printf("Size %d: %s\n\n\n", size, gold == res ? "SUCCESS!" : "FAILURE");

	// Then we repeat the above, with vectors of more irregular size
	size = 16 * 1024 + 220;

	// First we create a vector where neighbors have collisions
	vec = sequential_vector(size);
	for(int i = 0; i < size; i += 2){
		vec[i] = vec[i+1];
	}
	gold = size/2;
	res = test_count(vec, size, 1);
	printf("Expected: %d\n", gold);
	printf("Got:      %d\n", res);
	free(vec);
	printf("Size %d: %s\n\n\n", size, gold == res ? "SUCCESS!" : "FAILURE");

	// Then we create a vector where all elements are colliding
	vec = sequential_vector(size);
	for(int i = 0; i < size; i++){
		vec[i] = vec[0];
	}
	gold = size * (size - 1) / 2;
	res = test_count(vec, size, 1);
	printf("Expected: %d\n", gold);
	printf("Got:      %d\n", res);
	free(vec);
	printf("Size %d: %s\n\n\n", size, gold == res ? "SUCCESS!" : "FAILURE");

	// Finally, no collisions at all
	vec = sequential_vector(size);
	gold = 0;
	res = test_count(vec, size, 1);
	printf("Expected: %d\n", gold);
	printf("Got:      %d\n", res);
	free(vec);
	printf("Size %d: %s\n\n\n", size, gold == res ? "SUCCESS!" : "FAILURE");


	// Then we repeat the above, with a small vector
	size = 220;

	// First we create a vector where neighbors have collisions
	vec = sequential_vector(size);
	for(int i = 0; i < size; i += 2){
		vec[i] = vec[i+1];
	}
	gold = size/2;
	res = test_count(vec, size, 1);
	printf("Expected: %d\n", gold);
	printf("Got:      %d\n", res);
	free(vec);
	printf("Size %d: %s\n\n\n", size, gold == res ? "SUCCESS!" : "FAILURE");

	// Then we create a vector where all elements are colliding
	vec = sequential_vector(size);
	for(int i = 0; i < size; i++){
		vec[i] = vec[0];
	}
	gold = size * (size - 1) / 2;
	res = test_count(vec, size, 1);
	printf("Expected: %d\n", gold);
	printf("Got:      %d\n", res);
	free(vec);
	printf("Size %d: %s\n\n\n", size, gold == res ? "SUCCESS!" : "FAILURE");

	// Finally, no collisions at all
	vec = sequential_vector(size);
	gold = 0;
	res = test_count(vec, size, 1);
	printf("Expected: %d\n", gold);
	printf("Got:      %d\n", res);
	free(vec);
	printf("Size %d: %s\n\n\n", size, gold == res ? "SUCCESS!" : "FAILURE");
}


int main(int argc, char *argv[]){
	int vecSize = 32 * 16 * 1024;
	int iters   = 1;
	
	switch(argc){
		case 1:
			break;
		case 2:
			vecSize = atoi(argv[1]);
			break;
		case 3:
			vecSize = atoi(argv[1]);
			iters   = atoi(argv[2]);
			break;
		default:
			fprintf(stderr, "Usage: %s [problem_size] [no. iterations]\n", argv[0]);
			return 1;
	}
	
#if BUILD_TEST == 1
	t3();
#else
	t2(vecSize, iters);
#endif

	return 0;
}
