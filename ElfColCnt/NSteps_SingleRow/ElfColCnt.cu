#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#include "ElfColCnt.cuh"
#include "utils.h"

/* Multi-block reduce.
 * Accepts only vectors that are power of 2.
 */
__global__
void reduce(int *vec, int *result){
	extern __shared__ int sdata[];

	int tid = threadIdx.x;
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	sdata[tid] = vec[idx];
	__syncthreads();

	// Reduce
	for(int stride = blockDim.x >> 1; stride > 0; stride >>= 1){
		if(threadIdx.x < stride)
			sdata[threadIdx.x] += sdata[threadIdx.x+stride];

		__syncthreads();
	}

	result[blockIdx.x] = sdata[0];
}


/*
 * Collision Count procedure implemented in CUDA.
 *
 * This procedure parallelizes the sequential algorithm:
 * for i in 0:N-2
 *   for j in i+1:N-1
 *     collisions += (bead[i] == bead[j])
 * by performing just the outer 'for' in parallel.
 */
__global__
void count_collisions_cu(int3 *coords, int *result, int nCoords){
	int baseIdx = blockIdx.x * 1024;
	int horizontalId = threadIdx.x + blockIdx.x * blockDim.x;

	// Calculate number of iterations to execute
	// If we have 2048 nCoords and baseIdx is 0, we must execute 2048 iterations.
	int maxIterations = nCoords - baseIdx;

	// We read our element in a register (surplus threads will read anything)
	int3 buf = coords[horizontalId % nCoords];
	
	// Read the first block into shared memory (surplus threads read anything)
	extern __shared__ int3 sCoords[];
	sCoords[threadIdx.x] = coords[ (baseIdx + threadIdx.x) % nCoords ];
	__syncthreads();

	// Count collisions on first block, which is a problematic block
	int offset = 0;
	int collisions = 0;
	int iterations = 0;
	int limit = min(1024, maxIterations);
	for(; iterations < limit; iterations++){
		// Check collision
		int collision = (
			buf.x   == sCoords[offset].x
			& buf.y == sCoords[offset].y
			& buf.z == sCoords[offset].z
		);

		// Assert our comparison element is after the base element in 'buf'
		collision *= (baseIdx + offset > horizontalId);

		// Sum on global collisions
		collisions += collision;

		offset++;
	}

	baseIdx += 1024;
	offset  = 0;

	// Now do the rest of the blocks
	while(iterations < maxIterations){
		// Read 2 blocks. Modulus prevents invallid memory accesses.
		__syncthreads();
		sCoords[threadIdx.x] = coords[ (baseIdx + threadIdx.x) % nCoords ];
		sCoords[threadIdx.x + 1024] = coords[ (baseIdx + threadIdx.x + 1024) % nCoords ];
		__syncthreads();

		// Do 2048 iterations, or maybe less
		limit = min(iterations + 2048, maxIterations);
		for(; iterations < limit; iterations++){
			// Check collision
			collisions += (
				buf.x   == sCoords[offset].x
				& buf.y == sCoords[offset].y
				& buf.z == sCoords[offset].z
			);

			offset++;
		}
		
		baseIdx += 2048;
		offset  = 0;
	}

	// Sync before reducing collisions on shared memory
	__syncthreads();

	// Fill shared memory with collisions (surplus threads are ignored)
	extern __shared__ int sdata[];
	sdata[threadIdx.x] = collisions * (horizontalId < nCoords);
	__syncthreads();

	// Reduce 1024 elements
	for(int stride = 512; stride > 0; stride >>= 1){
		if(threadIdx.x < stride)
			sdata[threadIdx.x] += sdata[threadIdx.x+stride];

		__syncthreads();
	}
	
	// Export result
	if(threadIdx.x == 0){
		result[blockIdx.x] = sdata[0];
	}
}

/* Gets the next cuda stream in the circular list of streams.
 */
static
hipStream_t get_next_stream(){
	const int nStreams = 8;
	static hipStream_t streams[nStreams];
	static unsigned int launches = 0;

	// Allocate cuda streams in the first execution
	static int streamInit = 0;
	if(streamInit == 0){
		streamInit = 1;
		for(int i = 0; i < nStreams; i++){
			hipStreamCreate(&streams[i]);
		}
	}

	launches++;
	return streams[launches%nStreams];
}

/* Given a vector with 3D coordinates of points in the space,
 *   this function calculates the number of collisions among
 *   points, using CUDA-enable GPU.
 *
 * This functions just launches the kernel, returning a
 *   structure that can later be used to fetch the result
 *   back from the device memory.
 */
struct CollisionCountPromise
count_collisions_launch(int3 *vector, int size){
	if(size%2 != 0){
		fprintf(stderr, "Error: Vector size must be even.\n");
		exit(1);
	}

	int3 *d_vector;
	int *d_result;
	hipStream_t stream = get_next_stream();

	// Allocate cuda vector for the 3D coordinates
	hipMalloc(&d_vector, sizeof(int3) * size);
	hipMemcpyAsync(d_vector, vector, sizeof(int3) * size, hipMemcpyHostToDevice, stream);

	// Prepare kernel launch parameters
	const int elemInShmem = 2048; // 2048 because we need 2 blocks of 1024 elements in shmem.
	int nThreads = 1024;          // We allocate maximum number of threads per block.
	int nBlocks = divisionCeil(size, nThreads);
	int nShMem = elemInShmem * sizeof(int3); // Shared memory required

	// Allocate cuda memory for the number of collisions
	// This will also be used as a working vector for reducing among blocks
	int resultSize = higherEqualPow2(nBlocks);
	hipMalloc(&d_result, sizeof(int) * resultSize);
	hipMemsetAsync(d_result, 0, sizeof(int) * resultSize, stream); // Reset is needed due to size overestimation

	// Finally launch kernels
	count_collisions_cu<<<nBlocks, nThreads, nShMem, stream>>>(d_vector, d_result, size);
	
	// Reduce the result vector
	nBlocks = resultSize/1024;
	int workSize = resultSize;
	int *d_toReduce = d_result;
	int *d_reduced  = (int *) d_vector;
	while(true){
		if(nBlocks == 0){
			reduce<<<1, workSize, sizeof(int) * workSize, stream>>>(d_toReduce, d_reduced);
			break;
		}

		reduce<<<nBlocks, 1024, sizeof(int) * 1024, stream>>>(d_toReduce, d_reduced);

		// For the next run, vectors should be swapped
		int *aux = d_reduced;
		d_reduced = d_toReduce;
		d_toReduce = aux;

		// For the next run, the workSize and nBlocks are lower
		workSize = nBlocks;
		nBlocks = workSize/1024;
	}

	const struct CollisionCountPromise ret = { d_toReduce, d_reduced };
	return ret;
}

/* This procedure fetches the result from the call to the
 *   _launch correspondent.
 * The pointers within the promise structure are freed, so
 *   it shouldn't be used anywhere after a call to this function.
 */
int count_collisions_fetch(struct CollisionCountPromise promise){
	const int n = 1;
	int result[n];
	hipMemcpy(&result, promise.d_reduced, sizeof(int) * n, hipMemcpyDeviceToHost);

	hipFree(promise.d_toReduce);
	hipFree(promise.d_reduced);

	return result[0];
}
